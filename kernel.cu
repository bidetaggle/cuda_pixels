#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 16
#define KRANGE 8
#define TLEVEL 3

void prompt(int array[N][N]) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			if(array[i][j] < 10)
				printf("%i  ", array[i][j]);
			else if (array[i][j] < 100)
				printf("%i ", array[i][j]);
			else if (array[i][j] < 1000)
				printf("%i", array[i][j]);
			printf(" ");
		}
		printf("\n");
	}
	printf("\n");
}

__global__ void quantization(int A[N][N], int quantizationRange, int greyscaleRange) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N) {
		for (int t = 0; t<greyscaleRange / quantizationRange; t++)
			if (A[i][j] >= t*quantizationRange && A[i][j] < (t + 1)*(quantizationRange)) {
				A[i][j] = t;
				break;
			}
	}
}

__global__ void neigborsCount(int A[N][N]) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = 0;
	if (i < N && j < N) {
		/* edges */
		if (A[i - 1][j] != A[i][j] && (i-1) >= 0)
			k++;
		if (A[i + 1][j] != A[i][j] && (i+1) < N)
			k++;
		if (A[i][j - 1] != A[i][j] && (j-1) >= 0)
			k++;
		if (A[i][j + 1] != A[i][j] && (j+1) < N)
			k++;

		/* corners (diagonals) */
		if (A[i - 1][j - 1] != A[i][j] && (i - 1) >= 0 && (j - 1) >= 0)
			k++;
		if (A[i + 1][j - 1] != A[i][j] && (i + 1) < N && (j - 1) >= 0)
			k++;
		if (A[i - 1][j + 1] != A[i][j] && (i - 1) >= 0 && (j + 1) < N)
			k++;
		if (A[i + 1][j + 1] != A[i][j] && (i + 1) < N && (j + 1) < N)
			k++;

		A[i][j] = k;
	}
}

__global__ void initRand(unsigned int seed, hiprandState_t** states) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		i+j, /* the sequence number should be different for each core (unless you want all
					cores to get the same sequence of numbers for some reason - use thread id! */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		states[i][j]);
}

__global__ void randoms(hiprandState_t* states, int A[N][N]) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	A[i][j] = hiprand(&states[blockIdx.x]) % 100;
}

int main() {
	int A[N][N];

	hiprandState_t **states; //random states
	int(*d_A)[N]; //pointers to arrays of dimension N

	int greyscaleRange = int(pow(2, KRANGE));
	int quantizationRange = int(pow(2, KRANGE)) / int(pow(2, TLEVEL));

	printf("grayscale range : %i\n", greyscaleRange);
	printf("quantization range : %i\n", quantizationRange);
	printf("categories : %i\n", greyscaleRange / quantizationRange);

	// Kernel invocation
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

	hipMalloc((void**)&d_A, (N*N) * sizeof(int));
	hipMalloc((void**)&states, (N*N) * sizeof(hiprandState_t));
	initRand << <numBlocks, threadsPerBlock >> >(time(0), states);
	randoms << <numBlocks, threadsPerBlock >> >(states, d_A);
	hipMemcpy(A, d_A, (N*N) * sizeof(int), hipMemcpyDeviceToHost);
	
	/*
	//fill the array with random numbers
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i][j] = rand() % greyscaleRange;
		}
	}
	*/

	printf("\ninitial table : \n\n");
	prompt(A);

	//copying from host to device
	hipMemcpy(d_A, A, (N*N) * sizeof(int), hipMemcpyHostToDevice);

	/* quantization */
	quantization << <numBlocks, threadsPerBlock >> >(d_A, quantizationRange, greyscaleRange);
	hipMemcpy(A, (d_A), (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	printf("\nArray quantization : \n\n");
	prompt(A);

	/* count the neighbors */
	neigborsCount << <numBlocks, threadsPerBlock >> >(d_A);
	hipMemcpy(A, (d_A), (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	printf("\nDifferents neightbors count : \n\n");
	prompt(A);

	return 0;
}